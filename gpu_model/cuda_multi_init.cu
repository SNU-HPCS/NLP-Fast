#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "tensor_op_gpu.cuh"
#include "cuda_multi_init.cuh"
#include "bert_state.hpp"

int get_device_count(int *num_gpus) {
	hipError_t cuda_rc;
	if ((cuda_rc = hipGetDeviceCount(num_gpus)) != hipSuccess) {
		fprintf(stderr, "hipGetDeviceCount (cuda_rc: %d)\n", cuda_rc);
		return -1;
	}
	return 0;
}

int set_device(int gpu_id) {
	hipError_t cuda_rc;
	if ((cuda_rc = hipSetDevice(gpu_id)) != hipSuccess) {
		fprintf(stderr, "hipSetDevice (cuda_rc: %d)\n", cuda_rc);
		return -1;
	}
	if (gpu_id != 0) {
		if ((cuda_rc = hipDeviceEnablePeerAccess(0, 0)) != hipSuccess) {
			fprintf(stderr, "hipDeviceDisablePeerAccess (%d -> %d) (cuda_rc: %d)\n", gpu_id, 0, cuda_rc);
			return -1;
		}
		hipSetDevice(0);
		if ((cuda_rc = hipDeviceEnablePeerAccess(gpu_id, 0)) != hipSuccess) {
			fprintf(stderr, "hipDeviceDisablePeerAccess (%d -> %d) (cuda_rc: %d)\n", 0, gpu_id, cuda_rc);
			return -1;
		}
		hipSetDevice(gpu_id);
	}

	return 0;
}


int cuda_multi_cublas_init(Params *params, gpu_cuda_context_t *gpu_context, int gpu_id) {
	hipblasStatus_t cublas_rc;

	/// used for p2p memcpy
//	if (gpu_id == 0) {
//		gpu_context->streams = (hipStream_t *) malloc(sizeof(hipStream_t) * params->num_gpus);
//		for (int t_gpu_idx = 0; t_gpu_idx < params->num_gpus; t_gpu_idx++) {
//			hipStreamCreateWithFlags(&gpu_context->streams[t_gpu_idx], hipStreamNonBlocking);
//		}
//	}

	gpu_context->cublas_handles = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t) * 1);
	if ((cublas_rc = hipblasCreate(gpu_context->cublas_handles)) != HIPBLAS_STATUS_SUCCESS)  {
		fprintf(stderr, "(i:%d) hipblasCreate (cublas_rc: %d)\n", 0, cublas_rc);
		goto err;
	}

	return 0;
err:
	return -1;
}

int cuda_multi_host_mem_alloc(Params *params, BERT_State *bert_state, gpu_cuda_context_t *gpu_context) {
	const int num_batch = bert_state->num_batch;
	const int num_layer = bert_state->num_layer;
	const int num_heads = bert_state->num_heads;
	const int seq_length = bert_state->seq_length;
	const int hidden_size = bert_state->hidden_size;
	const int head_size = hidden_size / num_heads;
	const int feedforward_size = bert_state->feedforwardsize;
	const int ffw_chunk_size = feedforward_size / num_heads;
	hipError_t cuda_rc;

	/// Onevec & onemat
//	if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_onevec, 1 * hidden_size * sizeof(float))) != hipSuccess) {
//		fprintf(stderr, "hipHostMalloc (h_onevec) (cuda_rc: %d)\n", cuda_rc); goto err;
//	}
//	if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_onemat, hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
//		fprintf(stderr, "hipHostMalloc (h_onemat) (cuda_rc: %d)\n", cuda_rc); goto err;
//	}


	/// attention_mask
	gpu_context->h_attention_mask = (float **)malloc(num_batch * sizeof(float*));
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_attention_mask[batch_idx], seq_length * seq_length * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_attention_mask) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}

	/// Input
	gpu_context->h_input = (float**)malloc(num_batch * sizeof(float*));
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_input[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_input) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}

	/// Weight
	gpu_context->h_weight = (float ***)malloc(num_layer * sizeof(float**));
	gpu_context->h_weight_attention_fc_splitted = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->h_weight_attention_fc_bias_splitted = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->h_weight_ffw_prev = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->h_weight_ffw_prev_bias = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->h_weight_ffw_post_splitted = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->h_weight_ffw_post_bias_splitted = (float **)malloc(num_layer * sizeof(float*));
	for (int layer_idx = 0; layer_idx < num_layer; layer_idx++) {
		gpu_context->h_weight[layer_idx] = (float **)malloc(WEIGHT_MAX_NUM * sizeof(float*));
		memset(gpu_context->h_weight[layer_idx], 0, WEIGHT_MAX_NUM * sizeof(float*));

		/// Host memory
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_QLW], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_QLW]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_QLB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_QLB]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_KLW], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_KLW]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_KLB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_KLB]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_VLW], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_VLW]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_VLB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_VLB]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_ATTENTION]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_BIAS], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_ATTENTION_BIAS]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_GAMMA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_ATTENTION_GAMMA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_BETA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_ATTENTION_BETA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_PREV_FFW], hidden_size * feedforward_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_PREV_FFW]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_PREV_FFB], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_PREV_FFB]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_POST_FFW], feedforward_size * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_POST_FFW]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_POST_FFB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_POST_FFB]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_FF_GAMMA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_FF_GAMMA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight[layer_idx][WEIGHT_FF_BETA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight[WEIGHT_FF_BETA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight_attention_fc_splitted[layer_idx], num_heads * head_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight_attention_fc_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight_attention_fc_bias_splitted[layer_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight_attention_fc_bias_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}

		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight_ffw_prev[layer_idx], hidden_size * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight_ffw_prev) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight_ffw_prev_bias[layer_idx], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight_ffw_prev_bias) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight_ffw_post_splitted[layer_idx], num_heads * ffw_chunk_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight_ffw_post_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_weight_ffw_post_bias_splitted[layer_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_weight_ffw_post_bias_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}

	/// Intermediate buffer (Attention)
	gpu_context->h_buf_query   = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_key     = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_value   = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_score   = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_expsum  = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_softmax = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_att     = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_att_fc_result_split = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_att_layernorm = (float **)malloc(num_batch * sizeof(float**));
	gpu_context->h_buf_ffw_intermediate = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_ffw_gelu = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_ffw_result_split = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_ffw_layernorm = (float **)malloc(num_batch * sizeof(float**));
//	gpu_context->h_buf_layernorm_mean = (float **)malloc(num_batch * sizeof(float*));
//	gpu_context->h_buf_layernorm_tmp = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->h_buf_layernorm_nrm_v = (float **)malloc(num_batch * sizeof(float*));
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_query[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_query) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_key[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_key) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_value[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_value) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_score[batch_idx], num_heads * seq_length * seq_length * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_score) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_expsum[batch_idx], num_heads * seq_length * 1 * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_expsum) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_softmax[batch_idx], num_heads * seq_length * seq_length * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_softmax) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_att_fc_result_split[batch_idx], num_heads * seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_att_fc_result_split) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_ffw_intermediate[batch_idx], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_ffw_intermediate) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_ffw_gelu[batch_idx], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_ffw_gelu) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_ffw_result_split[batch_idx], num_heads * seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_ffw_result_split) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}

		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_att[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_att) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_att_layernorm[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_att_layernorm) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_ffw_layernorm[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_ffw_layernorm) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_layernorm_mean[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_buf_layernorm_mean) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
//		}
//		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_layernorm_tmp[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipHostMalloc (h_buf_layernorm_tmp) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
//		}
		if ((cuda_rc = hipHostMalloc((void**)&gpu_context->h_buf_layernorm_nrm_v[batch_idx], seq_length * 1 * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipHostMalloc (h_buf_layernorm_nrm_v) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
	}

	return 0;
err:
	return -1;
}

int cuda_multi_mem_alloc(Params *params, BERT_State *bert_state, gpu_cuda_context_t *gpu_context) {
	const int num_batch = bert_state->num_batch;
	const int num_layer = bert_state->num_layer;
	const int num_heads = bert_state->num_heads;
	const int seq_length = bert_state->seq_length;
	const int hidden_size = bert_state->hidden_size;
	const int head_size = hidden_size / num_heads;
	const int feedforward_size = bert_state->feedforwardsize;
	const int ffw_chunk_size = feedforward_size / num_heads;
	hipError_t cuda_rc;

	/// Onevec & onemat
//	if ((cuda_rc = hipMalloc((void**)&gpu_context->d_onevec, 1 * hidden_size * sizeof(float))) != hipSuccess) {
//		fprintf(stderr, "hipMalloc (d_onevec) (cuda_rc: %d)\n", cuda_rc); goto err;
//	}
//	if ((cuda_rc = hipMalloc((void**)&gpu_context->d_onemat, hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
//		fprintf(stderr, "hipMalloc (d_onemat) (cuda_rc: %d)\n", cuda_rc); goto err;
//	}


	/// attention_mask
	gpu_context->d_attention_mask = (float **)malloc(num_batch * sizeof(float*));
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_attention_mask[batch_idx], seq_length * seq_length * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_attention_mask) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}

	/// Input
	gpu_context->d_input = (float**)malloc(num_batch * sizeof(float*));
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_input[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_input) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}

	/// Weight
	gpu_context->d_weight = (float ***)malloc(num_layer * sizeof(float**));
	gpu_context->d_weight_attention_fc_splitted = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->d_weight_attention_fc_bias_splitted = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->d_weight_ffw_prev = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->d_weight_ffw_prev_bias = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->d_weight_ffw_post_splitted = (float **)malloc(num_layer * sizeof(float*));
	gpu_context->d_weight_ffw_post_bias_splitted = (float **)malloc(num_layer * sizeof(float*));
	for (int layer_idx = 0; layer_idx < num_layer; layer_idx++) {
		gpu_context->d_weight[layer_idx] = (float **)malloc(WEIGHT_MAX_NUM * sizeof(float*));
		memset(gpu_context->d_weight[layer_idx], 0, WEIGHT_MAX_NUM * sizeof(float*));

		/// GPU memory
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_QLW], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_QLW]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_QLB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_QLB]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_KLW], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_KLW]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_KLB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_KLB]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_VLW], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_VLW]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_VLB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_VLB]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION], hidden_size * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_ATTENTION]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION_BIAS], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_ATTENTION_BIAS]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION_GAMMA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_ATTENTION_GAMMA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION_BETA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_ATTENTION_BETA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_PREV_FFW], hidden_size * feedforward_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_PREV_FFW]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_PREV_FFB], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_PREV_FFB]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_POST_FFW], feedforward_size * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_POST_FFW]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_POST_FFB], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_POST_FFB]) (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_FF_GAMMA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_FF_GAMMA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight[layer_idx][WEIGHT_FF_BETA], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight[WEIGHT_FF_BETA]) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight_attention_fc_splitted[layer_idx], num_heads * head_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight_attention_fc_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight_attention_fc_bias_splitted[layer_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight_attention_fc_bias_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}

		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight_ffw_prev[layer_idx], hidden_size * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight_ffw_prev) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight_ffw_prev_bias[layer_idx], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight_ffw_prev_bias) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight_ffw_post_splitted[layer_idx], num_heads * ffw_chunk_size * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight_ffw_post_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_weight_ffw_post_bias_splitted[layer_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_weight_ffw_post_bias_splitted) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}

	/// Intermediate buffer (Attention)
	gpu_context->d_buf_query   = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_key     = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_value   = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_score   = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_expsum  = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_softmax = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_att     = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_att_fc_result_split = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_att_layernorm = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_ffw_intermediate = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_ffw_gelu = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_ffw_result_split = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_ffw_layernorm = (float **)malloc(num_batch * sizeof(float*));
//	gpu_context->d_buf_layernorm_mean = (float **)malloc(num_batch * sizeof(float*));
//	gpu_context->d_buf_layernorm_tmp = (float **)malloc(num_batch * sizeof(float*));
	gpu_context->d_buf_layernorm_nrm_v = (float **)malloc(num_batch * sizeof(float*));
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_query[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_query) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_key[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_key) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_value[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_value) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_score[batch_idx], num_heads * seq_length * seq_length * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_score) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_expsum[batch_idx], num_heads * seq_length * 1 * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_expsum) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_softmax[batch_idx], num_heads * seq_length * seq_length * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_softmax) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_att_fc_result_split[batch_idx], num_heads * seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_att_fc_result_split) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_ffw_intermediate[batch_idx], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_ffw_intermediate) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_ffw_gelu[batch_idx], seq_length * feedforward_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_ffw_gelu) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_ffw_result_split[batch_idx], num_heads * seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_ffw_result_split) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}

		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_att[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_att) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_att_layernorm[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_att_layernorm) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_ffw_layernorm[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_ffw_layernorm) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_layernorm_mean[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_buf_layernorm_mean) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
//		}
//		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_layernorm_tmp[batch_idx], seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//			fprintf(stderr, "hipMalloc (d_buf_layernorm_tmp) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
//		}
		if ((cuda_rc = hipMalloc((void**)&gpu_context->d_buf_layernorm_nrm_v[batch_idx], seq_length * 1 * sizeof(float))) != hipSuccess) {
			fprintf(stderr, "hipMalloc (d_buf_layernorm_nrm_v) (cuda_rc: %d) (reason: %s)\n", cuda_rc, hipGetErrorString(cuda_rc)); goto err;
		}
	}

	return 0;
err:
	return -1;
}

int cuda_multi_mem_init(Params *params, BERT_State *bert_state, gpu_cuda_context_t *gpu_context) {
	const int num_batch = bert_state->num_batch;
	const int num_layer = bert_state->num_layer;
	const int seq_length = bert_state->seq_length;
	const int hidden_size = bert_state->hidden_size;
	const int head_size = bert_state->hidden_size / bert_state->num_heads;
	const int ffw_chunk_size = bert_state->feedforwardsize / bert_state->num_heads;
	hipError_t cuda_rc;

	/// Setting host memory
	if (params->execution_mode == EXEC_MODE_VERIFICATION) {
		/// Attention Mask
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			matcopy_row_to_col_float(gpu_context->h_attention_mask[batch_idx], bert_state->m_attention_mask[batch_idx], seq_length, seq_length);
		}

		/// Input
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			matcopy_row_to_col_float(gpu_context->h_input[batch_idx], bert_state->embedding_output[batch_idx], seq_length, hidden_size);
		}

		/// Weight
		for (int layer_idx = 0; layer_idx < num_layer; layer_idx++) {
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_QLW], bert_state->weight[layer_idx][WEIGHT_QLW], hidden_size, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_QLB], bert_state->weight[layer_idx][WEIGHT_QLB], seq_length, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_KLW], bert_state->weight[layer_idx][WEIGHT_KLW], hidden_size, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_KLB], bert_state->weight[layer_idx][WEIGHT_KLB], seq_length, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_VLW], bert_state->weight[layer_idx][WEIGHT_VLW], hidden_size, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_VLB], bert_state->weight[layer_idx][WEIGHT_VLB], seq_length, hidden_size);
//			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION], bert_state->weight[layer_idx][WEIGHT_ATTENTION], hidden_size, hidden_size);
//			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_BIAS], bert_state->weight[layer_idx][WEIGHT_ATTENTION_BIAS], seq_length, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_GAMMA], bert_state->weight[layer_idx][WEIGHT_ATTENTION_GAMMA], seq_length, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_BETA], bert_state->weight[layer_idx][WEIGHT_ATTENTION_BETA], seq_length, hidden_size);
//			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_PREV_FFW], bert_state->weight[layer_idx][WEIGHT_PREV_FFW], hidden_size, feedforward_size);
//			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_PREV_FFB], bert_state->weight[layer_idx][WEIGHT_PREV_FFB], seq_length, feedforward_size);
//			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_POST_FFW], bert_state->weight[layer_idx][WEIGHT_POST_FFW], feedforward_size, hidden_size);
//			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_POST_FFB], bert_state->weight[layer_idx][WEIGHT_POST_FFB], seq_length, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_FF_GAMMA], bert_state->weight[layer_idx][WEIGHT_FF_GAMMA], seq_length, hidden_size);
			matcopy_row_to_col_float(gpu_context->h_weight[layer_idx][WEIGHT_FF_BETA], bert_state->weight[layer_idx][WEIGHT_FF_BETA], seq_length, hidden_size);

			for (int head_idx = 0; head_idx < bert_state->num_heads; head_idx++) {
				matcopy_row_to_col_float(&gpu_context->h_weight_attention_fc_splitted[layer_idx][head_idx * head_size * hidden_size],
				                         bert_state->weight_attention_fc_splitted[layer_idx][head_idx], head_size, hidden_size);
			}
			matcopy_row_to_col_float(gpu_context->h_weight_attention_fc_bias_splitted[layer_idx], bert_state->weight_attention_fc_bias_splitted[layer_idx], seq_length, hidden_size);

			for (int ffw_chunk_idx = 0; ffw_chunk_idx < bert_state->num_heads; ffw_chunk_idx++) {
				matcopy_row_to_col_float(&gpu_context->h_weight_ffw_prev[layer_idx][ffw_chunk_idx * hidden_size * ffw_chunk_size],
				                         bert_state->weight_ffw_prev_splitted[layer_idx][ffw_chunk_idx], hidden_size, ffw_chunk_size);
				matcopy_row_to_col_float(&gpu_context->h_weight_ffw_prev_bias[layer_idx][ffw_chunk_idx * seq_length * ffw_chunk_size],
				                         bert_state->weight_ffw_prev_bias_splitted[layer_idx][ffw_chunk_idx], seq_length, ffw_chunk_size);
				matcopy_row_to_col_float(&gpu_context->h_weight_ffw_post_splitted[layer_idx][ffw_chunk_idx * ffw_chunk_size * hidden_size],
				                         bert_state->weight_ffw_post_splitted[layer_idx][ffw_chunk_idx], ffw_chunk_size, hidden_size);
			}
			matcopy_row_to_col_float(gpu_context->h_weight_ffw_post_bias_splitted[layer_idx], bert_state->weight_ffw_post_bias_splitted[layer_idx], seq_length, hidden_size);
		}
	} else if (params->execution_mode == EXEC_MODE_PERF_TEST) {
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			/// We don't need transpose (because we assume data are random values)
			memcpy(gpu_context->h_attention_mask[batch_idx], bert_state->attention_mask[batch_idx], seq_length * seq_length * sizeof(float));
			memcpy(gpu_context->h_input[batch_idx], bert_state->embedding_output[batch_idx], seq_length * hidden_size * sizeof(float));
		}

		for (int layer_idx = 0; layer_idx < num_layer; layer_idx++) {
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_QLW], bert_state->weight[layer_idx][WEIGHT_QLW], hidden_size * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_QLB], bert_state->weight[layer_idx][WEIGHT_QLB], seq_length * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_KLW], bert_state->weight[layer_idx][WEIGHT_KLW], hidden_size * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_KLB], bert_state->weight[layer_idx][WEIGHT_KLB], seq_length * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_VLW], bert_state->weight[layer_idx][WEIGHT_VLW], hidden_size * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_VLB], bert_state->weight[layer_idx][WEIGHT_VLB], seq_length * hidden_size * sizeof(float));
//			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION], bert_state->weight[layer_idx][WEIGHT_ATTENTION], hidden_size * hidden_size * sizeof(float));
//			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_BIAS], bert_state->weight[layer_idx][WEIGHT_ATTENTION_BIAS], seq_length * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_GAMMA], bert_state->weight[layer_idx][WEIGHT_ATTENTION_GAMMA], seq_length * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_BETA], bert_state->weight[layer_idx][WEIGHT_ATTENTION_BETA], seq_length * hidden_size * sizeof(float));
//			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_PREV_FFW], bert_state->weight[layer_idx][WEIGHT_PREV_FFW], hidden_size * feedforward_size * sizeof(float));
//			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_PREV_FFB], bert_state->weight[layer_idx][WEIGHT_PREV_FFB], seq_length * feedforward_size * sizeof(float));
//			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_POST_FFW], bert_state->weight[layer_idx][WEIGHT_POST_FFW], feedforward_size * hidden_size * sizeof(float));
//			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_POST_FFB], bert_state->weight[layer_idx][WEIGHT_POST_FFB], seq_length * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_FF_GAMMA], bert_state->weight[layer_idx][WEIGHT_FF_GAMMA], seq_length * hidden_size * sizeof(float));
			memcpy(gpu_context->h_weight[layer_idx][WEIGHT_FF_BETA], bert_state->weight[layer_idx][WEIGHT_FF_BETA], seq_length * hidden_size * sizeof(float));

			for (int head_idx = 0; head_idx < bert_state->num_heads; head_idx++) {
				memcpy(&gpu_context->h_weight_attention_fc_splitted[layer_idx][head_idx * head_size * hidden_size],
				       bert_state->weight_attention_fc_splitted[layer_idx][head_idx], head_size * hidden_size * sizeof(float));
			}
			memcpy(gpu_context->h_weight_attention_fc_bias_splitted[layer_idx], bert_state->weight_attention_fc_bias_splitted[layer_idx], seq_length * hidden_size * sizeof(float));
			for (int ffw_chunk_idx = 0; ffw_chunk_idx < bert_state->num_heads; ffw_chunk_idx++) {
				memcpy(&gpu_context->h_weight_ffw_prev[layer_idx][ffw_chunk_idx * hidden_size * ffw_chunk_size],
				       bert_state->weight_ffw_prev_splitted[layer_idx][ffw_chunk_idx], hidden_size * ffw_chunk_size * sizeof(float));
				memcpy(&gpu_context->h_weight_ffw_prev_bias[layer_idx][ffw_chunk_idx * seq_length * ffw_chunk_size],
				       bert_state->weight_ffw_prev_bias_splitted[layer_idx][ffw_chunk_idx], seq_length * ffw_chunk_size * sizeof(float));
				memcpy(&gpu_context->h_weight_ffw_post_splitted[layer_idx][ffw_chunk_idx * ffw_chunk_size * hidden_size],
				       bert_state->weight_ffw_post_splitted[layer_idx][ffw_chunk_idx], ffw_chunk_size * hidden_size * sizeof(float));
			}
			memcpy(gpu_context->h_weight_ffw_post_bias_splitted[layer_idx], bert_state->weight_ffw_post_bias_splitted[layer_idx], seq_length * hidden_size * sizeof(float));
		}
	} else {
		assert(0);
	}

	/// Memcpy (host -> device)
	///  1. input values
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipMemcpy(gpu_context->d_input[batch_idx], gpu_context->h_input[batch_idx],
				seq_length * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "hipMemcpy (input) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}
	///  2. Attention mask
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		if ((cuda_rc = hipMemcpy(gpu_context->d_attention_mask[batch_idx], gpu_context->h_attention_mask[batch_idx],
				seq_length * seq_length * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "hipMemcpy (d_attention_mask) (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}

	/// 3. One vector & one matrix
//	for (int i = 0; i < 1 * hidden_size; i ++) { gpu_context->h_onevec[i] = 1.0f; }
//	if ((cuda_rc = hipMemcpy(gpu_context->d_onevec, gpu_context->h_onevec,
//			1 * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
//		fprintf(stderr, "hipMemcpy (d_onevec) (cuda_rc: %d)\n", cuda_rc); goto err;
//	}
//	for (int i = 0; i < hidden_size * hidden_size; i ++) { gpu_context->h_onemat[i] = 1.0f; }
//	if ((cuda_rc = hipMemcpy(gpu_context->d_onemat, gpu_context->h_onemat,
//			hidden_size * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
//		fprintf(stderr, "hipMemcpy (d_onemat) (cuda_rc: %d)\n", cuda_rc); goto err;
//	}
	hipDeviceSynchronize();

	return 0;
err:
	return -1;
}

static inline int cuda_layer_norm(int batch_idx, int thread_block_size,
		gpu_cuda_context_t *gpu_context, float *in, float *out, int M_dim, int N_dim,
		float *gamma_mat, float *beta_mat) {
	dim3 dimBlock(thread_block_size);
	dim3 dimGrid(M_dim * N_dim / thread_block_size);

	/// Calculate mean
	hipMemset(gpu_context->d_buf_layernorm_nrm_v[batch_idx], 0, M_dim * 1 * sizeof(float));
	g_layer_mean<<<dimGrid, dimBlock, 0>>>(
			in, gpu_context->d_buf_layernorm_nrm_v[batch_idx],
			M_dim, N_dim);
	g_layer_minus<<<dimGrid, dimBlock, 0>>>(
			in, gpu_context->d_buf_layernorm_nrm_v[batch_idx], in,
			M_dim, N_dim);

	/// Calculate norm2
	hipMemset(gpu_context->d_buf_layernorm_nrm_v[batch_idx], 0, M_dim * 1 * sizeof(float));
	g_layer_snrm2<<<dimGrid, dimBlock, 0>>>(
			in, gpu_context->d_buf_layernorm_nrm_v[batch_idx],
			M_dim, N_dim);
	g_sqrt<<<dimGrid, dimBlock, 0>>>(
			gpu_context->d_buf_layernorm_nrm_v[batch_idx], M_dim * 1);

	/// var calculation & gamma beta
	g_layer_norm_gamma_beta<<<dimGrid, dimBlock, 0>>>(
			in, out, gpu_context->d_buf_layernorm_nrm_v[batch_idx],
			gamma_mat, beta_mat,
			M_dim, N_dim);

	return 0;
//err:
//	return -1;
}
int cuda_multi_bert_main(multi_gpu_thread_arg_t* multi_gpu_arg) {
	const int gpu_id = multi_gpu_arg->gpu_id;
	Params *params = multi_gpu_arg->params;
	BERT_State *bert_state = multi_gpu_arg->bert_state;
	gpu_cuda_context_t *gpu_context = &multi_gpu_arg->gpu_contexts[gpu_id];
	gpu_cuda_context_t *gpu_context_gpu0 = &multi_gpu_arg->gpu_contexts[0];
	const int num_batch = bert_state->num_batch;
	const int num_layer = bert_state->num_layer;
	const int seq_length = bert_state->seq_length;
	const int hidden_size = bert_state->hidden_size;
	const int head_size = bert_state->hidden_size / bert_state->num_heads;
	const int ffw_chunk_size = bert_state->feedforwardsize / bert_state->num_heads;
	const int hidden_per_gpu = bert_state->hidden_size / multi_gpu_arg->gpu_num;
	const int ffw_per_gpu = bert_state->feedforwardsize / multi_gpu_arg->gpu_num;
	const int num_head_per_gpu = bert_state->num_heads / multi_gpu_arg->gpu_num;
	const int num_ffchunk_per_gpu = bert_state->num_heads / multi_gpu_arg->gpu_num;
	hipError_t cuda_rc;
	hipblasStatus_t cublas_rc;
	const float score_norm_factor = (1.0f / sqrtf((float)head_size));


	/// Assume d_input is already initialized
	for (int layer_idx = 0; layer_idx < num_layer; layer_idx++) {
		/////////////
		/// Q_GEN
		/////////////
		/// Q_GEN memcpy
		if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
			// copy weight_QLW
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight[layer_idx][WEIGHT_QLW][gpu_id * hidden_size * hidden_per_gpu],
					&gpu_context->h_weight[layer_idx][WEIGHT_QLW][gpu_id * hidden_size * hidden_per_gpu],
					hidden_size * hidden_per_gpu * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[Q_GEN] <QLW> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
			// copy weight_QLB
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight[layer_idx][WEIGHT_QLB][gpu_id * seq_length * hidden_per_gpu],
					&gpu_context->h_weight[layer_idx][WEIGHT_QLB][gpu_id * seq_length * hidden_per_gpu],
					seq_length * hidden_per_gpu * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[Q_GEN] <QLB> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
		}
		/// Q_GEN matmul & bias
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemm(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_N,
					seq_length, hidden_per_gpu, hidden_size,
					&alpha, gpu_context->d_input[batch_idx], seq_length,
					&gpu_context->d_weight[layer_idx][WEIGHT_QLW][gpu_id * hidden_size * hidden_per_gpu], hidden_size,
					&beta, &gpu_context->d_buf_query[batch_idx][gpu_id * seq_length * hidden_per_gpu], seq_length)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[Q_GEN] hipblasSgemm (cublas_rc: %d)\n", cublas_rc); goto err;
			}

			if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_per_gpu, &alpha,
					&gpu_context->d_weight[layer_idx][WEIGHT_QLB][gpu_id * seq_length * hidden_per_gpu], 1,
					&gpu_context->d_buf_query[batch_idx][gpu_id * seq_length * hidden_per_gpu], 1)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[Q_GEN] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}


		/////////////
		/// K_GEN
		/////////////
		/// K_GEN memcpy
		if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
			// copy weight_KLW
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight[layer_idx][WEIGHT_KLW][gpu_id * hidden_size * hidden_per_gpu],
					&gpu_context->h_weight[layer_idx][WEIGHT_KLW][gpu_id * hidden_size * hidden_per_gpu],
					hidden_size * hidden_per_gpu * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[K_GEN] <KLW> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
			// copy weight_KLB
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight[layer_idx][WEIGHT_KLB][gpu_id * seq_length * hidden_per_gpu],
					&gpu_context->h_weight[layer_idx][WEIGHT_KLB][gpu_id * seq_length * hidden_per_gpu],
					seq_length * hidden_per_gpu * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[K_GEN] <KLB> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
		}
		/// K_GEN matmul & bias
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemm(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_N,
					seq_length, hidden_per_gpu, hidden_size,
					&alpha, gpu_context->d_input[batch_idx], seq_length,
					&gpu_context->d_weight[layer_idx][WEIGHT_KLW][gpu_id * hidden_size * hidden_per_gpu], hidden_size,
					&beta, &gpu_context->d_buf_key[batch_idx][gpu_id * seq_length * hidden_per_gpu], seq_length)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[K_GEN] hipblasSgemm (cublas_rc: %d)\n", cublas_rc); goto err;
			}

			if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_per_gpu, &alpha,
					&gpu_context->d_weight[layer_idx][WEIGHT_KLB][gpu_id * seq_length * hidden_per_gpu], 1,
					&gpu_context->d_buf_key[batch_idx][gpu_id * seq_length * hidden_per_gpu], 1)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[K_GEN] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}


		/////////////
		/// Score_calculation
		/////////////
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemmStridedBatched(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_T,
					seq_length, seq_length, head_size,
					&alpha, &gpu_context->d_buf_query[batch_idx][seq_length * head_size * gpu_id * num_head_per_gpu],
					seq_length, seq_length * head_size,
					&gpu_context->d_buf_key[batch_idx][seq_length * head_size * gpu_id * num_head_per_gpu],
					seq_length, seq_length * head_size,
					&beta, &gpu_context->d_buf_score[batch_idx][seq_length * seq_length * gpu_id * num_head_per_gpu],
					seq_length, seq_length * seq_length, num_head_per_gpu)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[Score_calculation] hipblasSgemmStridedBatched (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}


		/////////////
		/// socre_norm & ATTENTION_LAYER_MASK_SUB
		/////////////
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			dim3 dimBlock(params->thread_block_size);
			dim3 dimGrid(num_head_per_gpu * seq_length * seq_length / params->thread_block_size);

			g_score_norm_layer_mask<<<dimGrid, dimBlock, 0>>>(
					&gpu_context->d_buf_score[batch_idx][gpu_id * num_head_per_gpu * seq_length * seq_length], score_norm_factor,
					gpu_context->d_attention_mask[batch_idx], seq_length, seq_length, num_head_per_gpu);
		}


		/////////////
		/// Softmax
		/////////////
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			dim3 dimBlock(params->thread_block_size);
			dim3 dimGrid(num_head_per_gpu * seq_length * seq_length / params->thread_block_size);

			/// exp_sum
			hipMemset(&gpu_context->d_buf_expsum[batch_idx][gpu_id * num_head_per_gpu * seq_length * 1],
					0, num_head_per_gpu * seq_length * 1 * sizeof(float));
			g_exp_sum<<<dimGrid, dimBlock, 0>>>(
					&gpu_context->d_buf_score[batch_idx][gpu_id * num_head_per_gpu * seq_length * seq_length],
					&gpu_context->d_buf_softmax[batch_idx][gpu_id * num_head_per_gpu * seq_length * seq_length],
					&gpu_context->d_buf_expsum[batch_idx][gpu_id * num_head_per_gpu * seq_length * 1],
					seq_length, seq_length, num_head_per_gpu);

			// div
			g_normalize<<<dimGrid, dimBlock, 0>>>(
					&gpu_context->d_buf_softmax[batch_idx][gpu_id * num_head_per_gpu * seq_length * seq_length],
					&gpu_context->d_buf_expsum[batch_idx][gpu_id * num_head_per_gpu * seq_length * 1],
					seq_length, seq_length, num_head_per_gpu);
		}


		/////////////
		/// V_GEN
		/////////////
		/// V_GEN memcpy
		if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
			// copy weight_VLW
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight[layer_idx][WEIGHT_VLW][gpu_id * hidden_size * hidden_per_gpu],
					&gpu_context->h_weight[layer_idx][WEIGHT_VLW][gpu_id * hidden_size * hidden_per_gpu],
					hidden_size * hidden_per_gpu * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[V_GEN] <VLW> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
			// copy weight_VLB
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight[layer_idx][WEIGHT_VLB][gpu_id * seq_length * hidden_per_gpu],
					&gpu_context->h_weight[layer_idx][WEIGHT_VLB][gpu_id * seq_length * hidden_per_gpu],
					seq_length * hidden_per_gpu * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[V_GEN] <VLB> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
		}
		/// V_GEN matmul & bias
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemm(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_N,
					seq_length, hidden_per_gpu, hidden_size,
					&alpha, gpu_context->d_input[batch_idx], seq_length,
					&gpu_context->d_weight[layer_idx][WEIGHT_VLW][gpu_id * hidden_size * hidden_per_gpu], hidden_size,
					&beta, &gpu_context->d_buf_value[batch_idx][gpu_id * seq_length * hidden_per_gpu], seq_length)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[V_GEN] hipblasSgemm (cublas_rc: %d)\n", cublas_rc); goto err;
			}

			if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_per_gpu, &alpha,
					&gpu_context->d_weight[layer_idx][WEIGHT_VLB][gpu_id * seq_length * hidden_per_gpu], 1,
					&gpu_context->d_buf_value[batch_idx][gpu_id * seq_length * hidden_per_gpu], 1)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[V_GEN] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}


		/////////////
		/// Weighted_Sum
		/////////////
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemmStridedBatched(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_N,
					seq_length, head_size, seq_length,
					&alpha, &gpu_context->d_buf_softmax[batch_idx][seq_length * seq_length * gpu_id * num_head_per_gpu],
					seq_length, seq_length * seq_length,
					&gpu_context->d_buf_value[batch_idx][seq_length * head_size * gpu_id * num_head_per_gpu],
					seq_length, seq_length * head_size,
					&beta, &gpu_context->d_buf_att[batch_idx][seq_length * head_size * gpu_id * num_head_per_gpu],
					seq_length, seq_length * head_size, num_head_per_gpu)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[Weighted_Sum] hipblasSgemmStridedBatched (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}


		/////////////
		/// ATTENTION_FC
		/////////////
		/// ATTENTION_FC memcpy
		if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
			// copy WEIGHT_ATTENTION
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight_attention_fc_splitted[layer_idx][gpu_id * hidden_per_gpu * hidden_size],
					&gpu_context->h_weight_attention_fc_splitted[layer_idx][gpu_id * hidden_per_gpu * hidden_size],
					hidden_per_gpu * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[ATTENTION_FC] <WEIGHT_ATTENTION> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
			if (gpu_id == 0) {
				// copy WEIGHT_ATTENTION_BIAS
				// cooperate between streams to load the bias value (since we use this bias values at reduce_sum only)
				if ((cuda_rc = hipMemcpy(
						&gpu_context->d_weight_attention_fc_bias_splitted[layer_idx][gpu_id * seq_length * hidden_size],
						&gpu_context->h_weight_attention_fc_bias_splitted[layer_idx][gpu_id * seq_length * hidden_size],
						seq_length * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
					fprintf(stderr, "[Feedforward_POST] <weight_FFW_POST_BIAS> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
					goto err;
				}
			}
		}

		/// ATTENTION_FC matmul & bias
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemmStridedBatched(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_N,
					seq_length, hidden_size, head_size,
					&alpha, &gpu_context->d_buf_att[batch_idx][seq_length * head_size * gpu_id * num_head_per_gpu],
					seq_length, seq_length * head_size,
					&gpu_context->d_weight_attention_fc_splitted[layer_idx][head_size * hidden_size * gpu_id * num_head_per_gpu],
					head_size, head_size * hidden_size,
					&beta, &gpu_context->d_buf_att_fc_result_split[batch_idx][seq_length * hidden_size * gpu_id * num_head_per_gpu],
					seq_length, seq_length * hidden_size, num_head_per_gpu)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[ATTENTION_FC] hipblasSgemmStridedBatched (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}


		/////////////
		/// ATTENTION_FC_Partial_Reduce_Sum (Since we partition it)
		/////////////
		for (int head_idx = (num_head_per_gpu * gpu_id + 1); head_idx < num_head_per_gpu * (gpu_id + 1); head_idx++) {
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				float alpha = 1.;
				if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
						&gpu_context->d_buf_att_fc_result_split[batch_idx][head_idx * seq_length * hidden_size], 1,
						&gpu_context->d_buf_att_fc_result_split[batch_idx][num_head_per_gpu * gpu_id * seq_length * hidden_size], 1)) != HIPBLAS_STATUS_SUCCESS) {
						fprintf(stderr, "[ATTENTION_FC_Reduce_Sum] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
				}
			}
		}


		///////////////////////////////////////////////
		/// Send partial results to GPU 0 (hipMemcpyPeer)
		if (gpu_id != 0) {
			if (params->memcpy_mode != MEMCPY_MODE_NO_ALL_OVERHEAD) {
				for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
					if ((cuda_rc = hipMemcpy(
							&gpu_context_gpu0->d_buf_att_fc_result_split[batch_idx][num_head_per_gpu * gpu_id * seq_length * hidden_size],
							&gpu_context->d_buf_att_fc_result_split[batch_idx][num_head_per_gpu * gpu_id * seq_length * hidden_size],
							seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToDevice)) != hipSuccess) {
						fprintf(stderr, "[Send partial results] d_buf_att_fc_result_split hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
						goto err;
					}
//					if ((cuda_rc = hipMemcpyPeer(
//							&gpu_context_gpu0->d_buf_att_fc_result_split[batch_idx][num_head_per_gpu * gpu_id * seq_length * hidden_size], 0,
//							&gpu_context->d_buf_att_fc_result_split[batch_idx][num_head_per_gpu * gpu_id * seq_length * hidden_size], gpu_id,
//							seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//						fprintf(stderr, "[Send partial results] d_buf_att_fc_result_split hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
//						goto err;
//					}
				}
			}
		}


		pthread_barrier_wait(multi_gpu_arg->multi_gpu_barrier_local_att_fc_rsum);
		if (gpu_id == 0) {
			/////////////
			/// ATTENTION_FC_Reduce_Sum (Since we partition it)
			/////////////
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				float alpha = 1.;
				for (int t_gpu_idx = 1; t_gpu_idx < params->num_gpus; t_gpu_idx++) {
					if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
							&gpu_context->d_buf_att_fc_result_split[batch_idx][num_head_per_gpu * t_gpu_idx * seq_length * hidden_size], 1,
							gpu_context->d_buf_att_fc_result_split[batch_idx], 1)) != HIPBLAS_STATUS_SUCCESS) {
						fprintf(stderr, "[ATTENTION_FC_Reduce_Sum] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
					}
				}

				if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
						gpu_context->d_weight_attention_fc_bias_splitted[layer_idx], 1,
						gpu_context->d_buf_att_fc_result_split[batch_idx], 1)) != HIPBLAS_STATUS_SUCCESS) {
					fprintf(stderr, "[ATTENTION_FC_Reduce_Sum] BIAS hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
				}
			}


			/////////////
			/// ATTENTION_RESIDUAL
			/////////////
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				float alpha = 1.;
				if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
						gpu_context->d_input[batch_idx], 1,
						gpu_context->d_buf_att_fc_result_split[batch_idx], 1)) != HIPBLAS_STATUS_SUCCESS) {
					fprintf(stderr, "[ATTENTION_RESIDUAL] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
				}
			}


			/////////////
			/// ATTENTION_NORM
			/////////////
			if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
				// copy WEIGHT_ATTENTION_GAMMA
				if ((cuda_rc = hipMemcpy(gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION_GAMMA],
						gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_GAMMA],
						seq_length * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
					fprintf(stderr, "[ATTENTION_NORM] <WEIGHT_ATTENTION_GAMMA> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
				}
				// copy WEIGHT_ATTENTION_BETA
				if ((cuda_rc = hipMemcpy(gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION_BETA],
						gpu_context->h_weight[layer_idx][WEIGHT_ATTENTION_BETA],
						seq_length * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
					fprintf(stderr, "[ATTENTION_NORM] <WEIGHT_ATTENTION_BETA> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
				}
			}
			/// Do calculation
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				cuda_layer_norm(batch_idx, params->thread_block_size,
								gpu_context, gpu_context->d_buf_att_fc_result_split[batch_idx], gpu_context->d_buf_att_layernorm[batch_idx],
								seq_length, hidden_size,
								gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION_GAMMA], gpu_context->d_weight[layer_idx][WEIGHT_ATTENTION_BETA]);
			}
		}


		/// Broadcast output to GPUs (hipMemcpyPeer)
		pthread_barrier_wait(multi_gpu_arg->multi_gpu_barrier_local_att_fc_rsum_rescopy);
		if (gpu_id != 0) {
			if (params->memcpy_mode != MEMCPY_MODE_NO_ALL_OVERHEAD) {
				for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
					if ((cuda_rc = hipMemcpy(
							&gpu_context->d_buf_att_layernorm[batch_idx][0],
							&gpu_context_gpu0->d_buf_att_layernorm[batch_idx][0],
							seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToDevice)) != hipSuccess) {
						fprintf(stderr, "[copy_res] <d_buf_att_layernorm> hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
						goto err;
					}
//					if ((cuda_rc = hipMemcpyPeer(
//							&gpu_context->d_buf_att_layernorm[batch_idx][0], gpu_id,
//							&gpu_context_gpu0->d_buf_att_layernorm[batch_idx][0], 0,
//							seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//						fprintf(stderr, "[copy_res] <d_buf_att_layernorm> hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
//						goto err;
//					}
				}
			}
		}
		pthread_barrier_wait(multi_gpu_arg->multi_gpu_barrier_local_att_fc_rsum_rescopy_done);
		///////////////////////////////////////////////


		/////////////
		/// Feedforward_PREV
		/////////////
		/// Feedforward_PREV memcpy
		if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
			// copy weight_FFW_PREV
			if ((cuda_rc = hipMemcpy(&gpu_context->d_weight_ffw_prev[layer_idx][gpu_id * hidden_size * ffw_per_gpu],
			                          &gpu_context->h_weight_ffw_prev[layer_idx][gpu_id * hidden_size * ffw_per_gpu],
			                          hidden_size * ffw_per_gpu * sizeof(float), hipMemcpyHostToDevice)) !=
			    hipSuccess) {
				fprintf(stderr, "[Feedforward_PREV] <weight_FFW_PREV> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
			// copy weight_FFW_PREV_BIAS
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight_ffw_prev_bias[layer_idx][gpu_id * seq_length * ffw_per_gpu],
					&gpu_context->h_weight_ffw_prev_bias[layer_idx][gpu_id * seq_length * ffw_per_gpu],
					seq_length * ffw_per_gpu * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[Feedforward_PREV] <weight_FFW_PREV_BIAS> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
		}
		/// Feedforward_PREV matmul & bias
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemm(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_N,
					seq_length, ffw_per_gpu, hidden_size,
					&alpha, gpu_context->d_buf_att_layernorm[batch_idx], seq_length,
					&gpu_context->d_weight_ffw_prev[layer_idx][gpu_id * hidden_size * ffw_per_gpu], hidden_size,
					&beta, &gpu_context->d_buf_ffw_intermediate[batch_idx][gpu_id * seq_length * ffw_per_gpu], seq_length)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[Feedforward_PREV] hipblasSgemm (cublas_rc: %d)\n", cublas_rc); goto err;
			}
			if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * ffw_per_gpu, &alpha,
					&gpu_context->d_weight_ffw_prev_bias[layer_idx][gpu_id * seq_length * ffw_per_gpu], 1,
					&gpu_context->d_buf_ffw_intermediate[batch_idx][gpu_id * seq_length * ffw_per_gpu], 1)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[Feedforward_PREV] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}


		/////////////
		/// Feedforward_GELU
		/////////////
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			dim3 dimBlock(params->thread_block_size);
			dim3 dimGrid(seq_length * ffw_per_gpu / params->thread_block_size);

			// exp
			g_gelu<<<dimGrid, dimBlock, 0>>>(
					&gpu_context->d_buf_ffw_intermediate[batch_idx][gpu_id * seq_length * ffw_per_gpu],
					&gpu_context->d_buf_ffw_gelu[batch_idx][gpu_id * seq_length * ffw_per_gpu],
					seq_length * ffw_per_gpu);
		}


		/////////////
		/// Feedforward_POST
		/////////////
		/// Feedforward_POST memcpy
		if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
			// copy weight_FFW_POST
			if ((cuda_rc = hipMemcpy(
					&gpu_context->d_weight_ffw_post_splitted[layer_idx][gpu_id * ffw_per_gpu * hidden_size],
					&gpu_context->h_weight_ffw_post_splitted[layer_idx][gpu_id * ffw_per_gpu * hidden_size],
					ffw_per_gpu * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "[Feedforward_POST] <weight_FFW_POST> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
				goto err;
			}
			if (gpu_id == 0) {
				// copy weight_FFW_POST_BIAS
				// cooperate between streams to load the bias value (since we use this bias values at reduce_sum only)
				if ((cuda_rc = hipMemcpy(
						&gpu_context->d_weight_ffw_post_bias_splitted[layer_idx][gpu_id * seq_length * hidden_size],
						&gpu_context->h_weight_ffw_post_bias_splitted[layer_idx][gpu_id * seq_length * hidden_size],
						seq_length * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
					fprintf(stderr, "[Feedforward_POST] <weight_FFW_POST_BIAS> hipMemcpy (cuda_rc: %d)\n", cuda_rc);
					goto err;
				}
			}
		}
		/// Feedforward_POST matmul & bias
		for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
			float alpha = 1., beta = 0.;
			if ((cublas_rc = hipblasSgemmStridedBatched(gpu_context->cublas_handles[0], HIPBLAS_OP_N, HIPBLAS_OP_N,
					seq_length, hidden_size, ffw_chunk_size,
					&alpha, &gpu_context->d_buf_ffw_gelu[batch_idx][gpu_id * seq_length * ffw_per_gpu],
					seq_length, seq_length * ffw_chunk_size,
					&gpu_context->d_weight_ffw_post_splitted[layer_idx][gpu_id * ffw_per_gpu * hidden_size],
					ffw_chunk_size, ffw_chunk_size * hidden_size,
					&beta, &gpu_context->d_buf_ffw_result_split[batch_idx][gpu_id * num_ffchunk_per_gpu * seq_length * hidden_size],
					seq_length, seq_length * hidden_size, num_ffchunk_per_gpu)) != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "[Feedforward_POST] hipblasSgemmStridedBatched (cublas_rc: %d)\n", cublas_rc); goto err;
			}
		}

		/////////////
		/// Feedforward_Partial_Reduce_Sum (Since we partition it)
		/////////////
		for (int ffw_chunk_idx = (num_ffchunk_per_gpu * gpu_id + 1); ffw_chunk_idx < num_ffchunk_per_gpu * (gpu_id + 1); ffw_chunk_idx++) {
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				float alpha = 1.;
				if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
						&gpu_context->d_buf_ffw_result_split[batch_idx][ffw_chunk_idx * seq_length * hidden_size], 1,
						&gpu_context->d_buf_ffw_result_split[batch_idx][num_ffchunk_per_gpu * gpu_id * seq_length * hidden_size], 1)) != HIPBLAS_STATUS_SUCCESS) {
					fprintf(stderr, "[Feedforward_Partial_Reduce_Sum] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
				}
			}
		}

		///////////////////////////////////////////////
		/// Send partial results to GPU 0 (hipMemcpyPeer)
		if (gpu_id != 0) {
			if (params->memcpy_mode != MEMCPY_MODE_NO_ALL_OVERHEAD) {
				for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
					if ((cuda_rc = hipMemcpy(
							&gpu_context_gpu0->d_buf_ffw_result_split[batch_idx][num_ffchunk_per_gpu * gpu_id * seq_length * hidden_size],
							&gpu_context->d_buf_ffw_result_split[batch_idx][num_ffchunk_per_gpu * gpu_id * seq_length * hidden_size],
							seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToDevice)) != hipSuccess) {
						fprintf(stderr, "[Send partial results] d_buf_ffw_result_split hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
						goto err;
					}
//					if ((cuda_rc = hipMemcpyPeer(
//							&gpu_context_gpu0->d_buf_ffw_result_split[batch_idx][num_ffchunk_per_gpu * gpu_id * seq_length * hidden_size], 0,
//							&gpu_context->d_buf_ffw_result_split[batch_idx][num_ffchunk_per_gpu * gpu_id * seq_length * hidden_size], gpu_id,
//							seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//						fprintf(stderr, "[Send partial results] d_buf_ffw_result_split hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
//						goto err;
//					}
				}
			}
		}


		pthread_barrier_wait(multi_gpu_arg->multi_gpu_barrier_local_ffw_rsum);
		if (gpu_id == 0) {
			/////////////
			/// Feedforward_Reduce_Sum (Since we partition it)
			/////////////
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				float alpha = 1.;
				for (int t_gpu_idx = 1; t_gpu_idx < params->num_gpus; t_gpu_idx++) {
					if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
							&gpu_context->d_buf_ffw_result_split[batch_idx][num_ffchunk_per_gpu * t_gpu_idx * seq_length * hidden_size], 1,
							gpu_context->d_buf_ffw_result_split[batch_idx], 1)) != HIPBLAS_STATUS_SUCCESS) {
						fprintf(stderr, "[Feedforward_Reduce_Sum] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
					}
				}
				if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
						gpu_context->d_weight_ffw_post_bias_splitted[layer_idx], 1,
						gpu_context->d_buf_ffw_result_split[batch_idx], 1)) != HIPBLAS_STATUS_SUCCESS) {
					fprintf(stderr, "[Feedforward_Reduce_Sum] BIAS hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
				}
			}


			/////////////
			/// Feedforward_RESIDUAL
			/////////////
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				float alpha = 1.;
				if ((cublas_rc = hipblasSaxpy(gpu_context->cublas_handles[0], seq_length * hidden_size, &alpha,
						gpu_context->d_buf_att_layernorm[batch_idx], 1,
						gpu_context->d_buf_ffw_result_split[batch_idx], 1)) != HIPBLAS_STATUS_SUCCESS) {
					fprintf(stderr, "[ATTENTION_RESIDUAL] hipblasSaxpy (cublas_rc: %d)\n", cublas_rc); goto err;
				}
			}


			/////////////
			/// Feedforward_NORM
			/////////////
			if (params->memcpy_mode == MEMCPY_MODE_ALL_OVERHEAD) {
				// copy WEIGHT_FF_GAMMA
				if ((cuda_rc = hipMemcpy(gpu_context->d_weight[layer_idx][WEIGHT_FF_GAMMA],
						gpu_context->h_weight[layer_idx][WEIGHT_FF_GAMMA],
						seq_length * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
					fprintf(stderr, "[Feedforward_NORM] <WEIGHT_FF_GAMMA> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
				}
				// copy WEIGHT_FF_BETA
				if ((cuda_rc = hipMemcpy(gpu_context->d_weight[layer_idx][WEIGHT_FF_BETA],
						gpu_context->h_weight[layer_idx][WEIGHT_FF_BETA],
						seq_length * hidden_size * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
					fprintf(stderr, "[Feedforward_NORM] <WEIGHT_FF_BETA> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
				}
			}
			/// Do calculation
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				cuda_layer_norm(batch_idx, params->thread_block_size,
								gpu_context, gpu_context->d_buf_ffw_result_split[batch_idx], gpu_context->d_buf_ffw_layernorm[batch_idx],
								seq_length, hidden_size,
								gpu_context->d_weight[layer_idx][WEIGHT_FF_GAMMA], gpu_context->d_weight[layer_idx][WEIGHT_FF_BETA]);
			}
		}


		/// Broadcast output to GPUs (hipMemcpyPeer)
		pthread_barrier_wait(multi_gpu_arg->multi_gpu_barrier_local_ffw_rsum_rescopy);
		if (params->memcpy_mode != MEMCPY_MODE_NO_ALL_OVERHEAD) {
			for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
				if ((cuda_rc = hipMemcpy(gpu_context->d_input[batch_idx],
										  gpu_context_gpu0->d_buf_ffw_layernorm[batch_idx],
										  seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToDevice)) != hipSuccess) {
					fprintf(stderr, "[copy_res] <d_buf_att_layernorm> hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
					goto err;
				}
//				if ((cuda_rc = hipMemcpyPeer(gpu_context->d_input[batch_idx], gpu_id,
//											  gpu_context_gpu0->d_buf_ffw_layernorm[batch_idx], 0,
//											  seq_length * hidden_size * sizeof(float))) != hipSuccess) {
//					fprintf(stderr, "[copy_res] <d_buf_att_layernorm> hipMemcpyPeer (cuda_rc: %d)\n", cuda_rc);
//					goto err;
//				}
			}
		}
		pthread_barrier_wait(multi_gpu_arg->multi_gpu_barrier_local_ffw_rsum_rescopy_done);
		///////////////////////////////////////////////
	}

	return 0;
err:
	return -1;
}

void dump_gpu_matrix(float *gpu_mem, int M, int N, int gpu_id, const char *prefix_msg) {
	hipError_t cuda_rc;
	float *host_buf;
	char str_buf[4096];

	if ((cuda_rc = hipHostMalloc((void**)&host_buf, M * N * sizeof(float))) != hipSuccess) {
		fprintf(stderr, "<dump_gpu_matrix> Fail to hipHostMalloc (reason: %s)\n", hipGetErrorString(cuda_rc));
		assert(0);
	}

	if ((cuda_rc = hipMemcpy(host_buf, gpu_mem, M * N * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "<dump_gpu_matrix> hipMemcpy (cuda_rc: %d)\n", cuda_rc);\
		assert(0);
	}

	snprintf(str_buf, sizeof(str_buf), "<%s> [gpu_id: %d] (M: %d, N: %d)\n"
									   "    %+-4.6f %+-4.6f ... %+-4.6f %+-4.6f\n"
									   "    %+-4.6f %+-4.6f ... %+-4.6f %+-4.6f\n"
									   "    ...\n"
									   "    %+-4.6f %+-4.6f ... %+-4.6f %+-4.6f\n"
									   "    %+-4.6f %+-4.6f ... %+-4.6f %+-4.6f\n",
									   prefix_msg, gpu_id, M, N,
									   host_buf[0],   host_buf[M],	   host_buf[(N-2)*M],     host_buf[(N-1)*M],
									   host_buf[1],   host_buf[M+1],   host_buf[(N-2)*M+1],   host_buf[(N-1)*M+1],
									   host_buf[M-2], host_buf[M+M-2], host_buf[(N-2)*M+M-2], host_buf[(N-1)*M+M-2],
									   host_buf[M-1], host_buf[M+M-1], host_buf[(N-2)*M+M-1], host_buf[(N-1)*M+M-1]);
	printf("%s", str_buf);
	hipFree(host_buf);
}

void sync_all_buf_to_host(BERT_State *bert_state, gpu_cuda_context_t *gpu_context) {
	const int num_batch = bert_state->num_batch;
	const int num_heads = bert_state->num_heads;
	const int seq_length = bert_state->seq_length;
	const int hidden_size = bert_state->hidden_size;
	const int feedforward_size = bert_state->feedforwardsize;
//	const int head_size = bert_state->hidden_size / bert_state->num_heads;
//	const int ffw_chunk_size = bert_state->feedforwardsize / bert_state->num_heads;
	hipError_t cuda_rc;

	hipDeviceSynchronize();
	for (int batch_idx = 0; batch_idx < num_batch; batch_idx++) {
		// copy d_input
		if ((cuda_rc = hipMemcpy(gpu_context->h_input[batch_idx], gpu_context->d_input[batch_idx],
				seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_input> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_query
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_query[batch_idx], gpu_context->d_buf_query[batch_idx],
		                          seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_query> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_key
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_key[batch_idx], gpu_context->d_buf_key[batch_idx],
		                          seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_key> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_value
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_value[batch_idx], gpu_context->d_buf_value[batch_idx],
		                          seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_value> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_score
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_score[batch_idx], gpu_context->d_buf_score[batch_idx],
		                          num_heads * seq_length * seq_length * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_score> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_expsum
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_expsum[batch_idx], gpu_context->d_buf_expsum[batch_idx],
		                          num_heads * seq_length * 1 * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_expsum> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_softmax
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_softmax[batch_idx], gpu_context->d_buf_softmax[batch_idx],
		                          num_heads * seq_length * seq_length * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_softmax> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_att
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_att[batch_idx], gpu_context->d_buf_att[batch_idx],
				seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_att> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_att_fc_result_split
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_att_fc_result_split[batch_idx], gpu_context->d_buf_att_fc_result_split[batch_idx],
		                          num_heads * seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_att_fc_result_split> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_att_layernorm
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_att_layernorm[batch_idx], gpu_context->d_buf_att_layernorm[batch_idx],
				seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_att_layernorm> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}

		// copy d_buf_ffw_intermediate
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_ffw_intermediate[batch_idx], gpu_context->d_buf_ffw_intermediate[batch_idx],
		                          seq_length * feedforward_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_ffw_intermediate> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_ffw_gelu
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_ffw_gelu[batch_idx], gpu_context->d_buf_ffw_gelu[batch_idx],
		                          seq_length * feedforward_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_ffw_gelu> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_ffw_result_split
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_ffw_result_split[batch_idx], gpu_context->d_buf_ffw_result_split[batch_idx],
		                          num_heads * seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_ffw_result_split> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
		// copy d_buf_ffw_layernorm
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_ffw_layernorm[batch_idx], gpu_context->d_buf_ffw_layernorm[batch_idx],
		                          seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_ffw_layernorm> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
//		// copy d_buf_layernorm_mean
//		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_layernorm_mean[batch_idx], gpu_context->d_buf_layernorm_mean[batch_idx],
//				seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
//			fprintf(stderr, "[SYNC_DEBUG] <d_buf_layernorm_mean> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
//		// copy d_buf_layernorm_tmp
//		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_layernorm_tmp[batch_idx], gpu_context->d_buf_layernorm_tmp[batch_idx],
//		                          seq_length * hidden_size * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
//			fprintf(stderr, "[SYNC_DEBUG] <d_buf_layernorm_tmp> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
//		}
		// copy d_buf_layernorm_nrm_v
		if ((cuda_rc = hipMemcpy(gpu_context->h_buf_layernorm_nrm_v[batch_idx], gpu_context->d_buf_layernorm_nrm_v[batch_idx],
				seq_length * 1 * sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
			fprintf(stderr, "[SYNC_DEBUG] <d_buf_layernorm_nrm_v> hipMemcpy (cuda_rc: %d)\n", cuda_rc); goto err;
		}
	}
	hipDeviceSynchronize();
	printf("[SYNC_DEBUG] buffer sync done\n");

	return;
err:
	fprintf(stderr, "Fail to hipMemcpy (reason: %s)\n", hipGetErrorString(cuda_rc));
	assert(0);
}


void cuda_multi_host_context_deinit(BERT_State *bert_state, gpu_cuda_context_t *gpu_context) {
	/// Host memory
//	hipHostFree(gpu_context->h_onevec); gpu_context->h_onevec = nullptr;
//	hipHostFree(gpu_context->h_onemat); gpu_context->h_onemat = nullptr;

	if (gpu_context->h_attention_mask) {
		for (int batch_idx = 0; batch_idx < bert_state->num_batch; batch_idx++) {
			if (gpu_context->h_attention_mask[batch_idx]) hipHostFree(gpu_context->h_attention_mask[batch_idx]);
		}
		free(gpu_context->h_attention_mask); gpu_context->h_attention_mask = nullptr;
	}

	if (gpu_context->h_input) {
		for (int batch_idx = 0; batch_idx < bert_state->num_batch; batch_idx++) {
			if (gpu_context->h_input[batch_idx]) hipHostFree(gpu_context->h_input[batch_idx]);
		}
		free(gpu_context->h_input); gpu_context->h_input = nullptr;
	}

	if (gpu_context->h_weight) {
		for (int layer_idx = 0; layer_idx < bert_state->num_layer; layer_idx++) {
			for (int weight_idx = 0; weight_idx < WEIGHT_MAX_NUM; weight_idx++) {
				if (gpu_context->h_weight[layer_idx][weight_idx]) hipHostFree(gpu_context->h_weight[layer_idx][weight_idx]);
			}
			free(gpu_context->h_weight[layer_idx]);

			hipHostFree(gpu_context->h_weight_attention_fc_splitted[layer_idx]);
			hipHostFree(gpu_context->h_weight_attention_fc_bias_splitted[layer_idx]);

			hipHostFree(gpu_context->h_weight_ffw_prev[layer_idx]);
			hipHostFree(gpu_context->h_weight_ffw_prev_bias[layer_idx]);
			hipHostFree(gpu_context->h_weight_ffw_post_splitted[layer_idx]);
			hipHostFree(gpu_context->h_weight_ffw_post_bias_splitted[layer_idx]);
		}
		free(gpu_context->h_weight); gpu_context->h_weight = nullptr;
		free(gpu_context->h_weight_attention_fc_splitted); gpu_context->h_weight_attention_fc_splitted = nullptr;
		free(gpu_context->h_weight_attention_fc_bias_splitted); gpu_context->h_weight_attention_fc_bias_splitted = nullptr;
		free(gpu_context->h_weight_ffw_prev); gpu_context->h_weight_ffw_prev = nullptr;
		free(gpu_context->h_weight_ffw_prev_bias); gpu_context->h_weight_ffw_prev_bias = nullptr;
		free(gpu_context->h_weight_ffw_post_splitted); gpu_context->h_weight_ffw_post_splitted = nullptr;
		free(gpu_context->h_weight_ffw_post_bias_splitted); gpu_context->h_weight_ffw_post_bias_splitted = nullptr;
	}

	for (int batch_idx = 0; batch_idx < bert_state->num_batch; batch_idx++) {
		hipHostFree(gpu_context->h_buf_query[batch_idx]);
		hipHostFree(gpu_context->h_buf_key[batch_idx]);
		hipHostFree(gpu_context->h_buf_value[batch_idx]);
		hipHostFree(gpu_context->h_buf_score[batch_idx]);
		hipHostFree(gpu_context->h_buf_expsum[batch_idx]);
		hipHostFree(gpu_context->h_buf_softmax[batch_idx]);
		hipHostFree(gpu_context->h_buf_att_fc_result_split[batch_idx]);
		hipHostFree(gpu_context->h_buf_ffw_intermediate[batch_idx]);
		hipHostFree(gpu_context->h_buf_ffw_gelu[batch_idx]);
		hipHostFree(gpu_context->h_buf_ffw_result_split[batch_idx]);

		hipHostFree(gpu_context->h_buf_att[batch_idx]);
		hipHostFree(gpu_context->h_buf_att_layernorm[batch_idx]);
		hipHostFree(gpu_context->h_buf_ffw_layernorm[batch_idx]);
//		hipHostFree(gpu_context->h_buf_layernorm_mean[batch_idx]);
//		hipHostFree(gpu_context->h_buf_layernorm_tmp[batch_idx]);
		hipHostFree(gpu_context->h_buf_layernorm_nrm_v[batch_idx]);
	}
	free(gpu_context->h_buf_query); gpu_context->h_buf_query = nullptr;
	free(gpu_context->h_buf_key); gpu_context->h_buf_key = nullptr;
	free(gpu_context->h_buf_value); gpu_context->h_buf_value = nullptr;
	free(gpu_context->h_buf_score); gpu_context->h_buf_score = nullptr;
	free(gpu_context->h_buf_expsum); gpu_context->h_buf_expsum = nullptr;
	free(gpu_context->h_buf_softmax); gpu_context->h_buf_softmax = nullptr;
	free(gpu_context->h_buf_att); gpu_context->h_buf_att = nullptr;
	free(gpu_context->h_buf_att_fc_result_split); gpu_context->h_buf_att_fc_result_split = nullptr;
	free(gpu_context->h_buf_att_layernorm); gpu_context->h_buf_att_layernorm = nullptr;
	free(gpu_context->h_buf_ffw_intermediate); gpu_context->h_buf_ffw_intermediate = nullptr;
	free(gpu_context->h_buf_ffw_gelu); gpu_context->h_buf_ffw_gelu = nullptr;
	free(gpu_context->h_buf_ffw_result_split); gpu_context->h_buf_ffw_result_split = nullptr;
	free(gpu_context->h_buf_ffw_layernorm); gpu_context->h_buf_att_layernorm = nullptr;
//	free(gpu_context->h_buf_layernorm_mean); gpu_context->h_buf_layernorm_mean = nullptr;
//	free(gpu_context->h_buf_layernorm_tmp); gpu_context->h_buf_layernorm_tmp = nullptr;
	free(gpu_context->h_buf_layernorm_nrm_v); gpu_context->h_buf_layernorm_nrm_v = nullptr;
}

void cuda_multi_dev_context_deinit(BERT_State *bert_state, gpu_cuda_context_t *gpu_context) {
	// Destroy CUDA streams, cublas contexts
	hipblasDestroy(gpu_context->cublas_handles[0]);
	free(gpu_context->cublas_handles);

	/// GPU memory
//	hipFree(gpu_context->d_onevec); gpu_context->d_onevec = nullptr;
//	hipFree(gpu_context->d_onemat); gpu_context->d_onemat = nullptr;

	if (gpu_context->d_attention_mask) {
		for (int batch_idx = 0; batch_idx < bert_state->num_batch; batch_idx++) {
			if (gpu_context->d_attention_mask[batch_idx]) hipFree(gpu_context->d_attention_mask[batch_idx]);
		}
		free(gpu_context->d_attention_mask); gpu_context->d_attention_mask = nullptr;
	}

	if (gpu_context->d_input) {
		for (int batch_idx = 0; batch_idx < bert_state->num_batch; batch_idx++) {
			if (gpu_context->d_input[batch_idx]) hipFree(gpu_context->d_input[batch_idx]);
		}
		free(gpu_context->d_input); gpu_context->d_input = nullptr;
	}

	if (gpu_context->d_weight) {
		for (int layer_idx = 0; layer_idx < bert_state->num_layer; layer_idx++) {
			for (int weight_idx = 0; weight_idx < WEIGHT_MAX_NUM; weight_idx++) {
				if (gpu_context->d_weight[layer_idx][weight_idx]) hipFree(gpu_context->d_weight[layer_idx][weight_idx]);
			}
			free(gpu_context->d_weight[layer_idx]);

			hipFree(gpu_context->d_weight_attention_fc_splitted[layer_idx]);
			hipFree(gpu_context->d_weight_attention_fc_bias_splitted[layer_idx]);

			hipFree(gpu_context->d_weight_ffw_prev[layer_idx]);
			hipFree(gpu_context->d_weight_ffw_prev_bias[layer_idx]);
			hipFree(gpu_context->d_weight_ffw_post_splitted[layer_idx]);
			hipFree(gpu_context->d_weight_ffw_post_bias_splitted[layer_idx]);
		}
		free(gpu_context->d_weight); gpu_context->d_weight = nullptr;
		free(gpu_context->d_weight_attention_fc_splitted); gpu_context->d_weight_attention_fc_splitted = nullptr;
		free(gpu_context->d_weight_attention_fc_bias_splitted); gpu_context->d_weight_attention_fc_bias_splitted = nullptr;
		free(gpu_context->d_weight_ffw_prev); gpu_context->d_weight_ffw_prev = nullptr;
		free(gpu_context->d_weight_ffw_prev_bias); gpu_context->d_weight_ffw_prev_bias = nullptr;
		free(gpu_context->d_weight_ffw_post_splitted); gpu_context->h_weight_ffw_post_splitted = nullptr;
		free(gpu_context->d_weight_ffw_post_bias_splitted); gpu_context->h_weight_ffw_post_bias_splitted = nullptr;
	}

	for (int batch_idx = 0; batch_idx < bert_state->num_batch; batch_idx++) {
		hipFree(gpu_context->d_buf_query[batch_idx]);
		hipFree(gpu_context->d_buf_key[batch_idx]);
		hipFree(gpu_context->d_buf_value[batch_idx]);
		hipFree(gpu_context->d_buf_score[batch_idx]);
		hipFree(gpu_context->d_buf_expsum[batch_idx]);
		hipFree(gpu_context->d_buf_softmax[batch_idx]);
		hipFree(gpu_context->d_buf_att_fc_result_split[batch_idx]);
		hipFree(gpu_context->d_buf_ffw_intermediate[batch_idx]);
		hipFree(gpu_context->d_buf_ffw_gelu[batch_idx]);
		hipFree(gpu_context->d_buf_ffw_result_split[batch_idx]);

		hipFree(gpu_context->d_buf_att[batch_idx]);
		hipFree(gpu_context->d_buf_att_layernorm[batch_idx]);
		hipFree(gpu_context->d_buf_ffw_layernorm[batch_idx]);
//		hipFree(gpu_context->d_buf_layernorm_mean[batch_idx]);
//		hipFree(gpu_context->d_buf_layernorm_tmp[batch_idx]);
		hipFree(gpu_context->d_buf_layernorm_nrm_v[batch_idx]);
	}
	free(gpu_context->d_buf_query); gpu_context->d_buf_query = nullptr;
	free(gpu_context->d_buf_key); gpu_context->d_buf_key = nullptr;
	free(gpu_context->d_buf_value); gpu_context->d_buf_value = nullptr;
	free(gpu_context->d_buf_score); gpu_context->d_buf_score = nullptr;
	free(gpu_context->d_buf_expsum); gpu_context->d_buf_expsum = nullptr;
	free(gpu_context->d_buf_softmax); gpu_context->d_buf_softmax = nullptr;
	free(gpu_context->d_buf_att); gpu_context->d_buf_att = nullptr;
	free(gpu_context->d_buf_att_fc_result_split); gpu_context->d_buf_att_fc_result_split = nullptr;
	free(gpu_context->d_buf_att_layernorm); gpu_context->d_buf_att_layernorm = nullptr;
	free(gpu_context->d_buf_ffw_intermediate); gpu_context->d_buf_ffw_intermediate = nullptr;
	free(gpu_context->d_buf_ffw_gelu); gpu_context->d_buf_ffw_gelu = nullptr;
	free(gpu_context->d_buf_ffw_result_split); gpu_context->d_buf_ffw_result_split = nullptr;
	free(gpu_context->d_buf_ffw_layernorm); gpu_context->d_buf_ffw_layernorm = nullptr;
//	free(gpu_context->d_buf_layernorm_mean); gpu_context->d_buf_layernorm_mean = nullptr;
//	free(gpu_context->d_buf_layernorm_tmp); gpu_context->d_buf_layernorm_tmp = nullptr;
	free(gpu_context->d_buf_layernorm_nrm_v); gpu_context->d_buf_layernorm_nrm_v = nullptr;
}